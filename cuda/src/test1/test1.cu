

#include <hip/hip_runtime.h>
#include <iostream>
#include <cuda_device_runtime_api.h>

int main(int argc, char**argv)
{
    int count = 0;
    auto ec = hipGetDeviceCount(&count);
    if (ec != hipSuccess)
    {
        std::cout << "Failed get cuda device count, error code is "
            << ec << std::endl;
    }
    else
    {
        std::cout << "Got " << count << " cuda devices" << std::endl;
    }

    return 0;
}
